#include "hip/hip_runtime.h"
#include "gpu_louvain.h"

#include <algorithm>
#include <stdint.h>
#include <string.h>
#include <thrust/reduce.h>
#include <thrust/sort.h>
#include <thrust/scan.h>

#include "utils.h"

#define BLOCKS 80
#define THREADS_PER_BLOCK 1024
#define EPS (1e-12)

uint32_t ARRAY_SIZE = 1LL << 30;

__global__ void
prepare_data_structures_kernel(int N, int E, Edge* edges, int* c, float* k, int* nodes_comm, int* new_nodes_comm)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int num_threads = blockDim.x * gridDim.x;
    for (int i = tid; i < N; i += num_threads)
    {
        c[i] = i;
    }
    for (int i = tid; i < E; i += num_threads)
    {
        atomicAdd(&k[edges[i].dst], edges[i].weight);
    }

    for (int i = tid; i < N; i += num_threads)
    {
        nodes_comm[i] = 1;
    }
    for (int i = tid; i < N; i += num_threads)
    {
        new_nodes_comm[i] = 1;
    }
}

__host__ void prepare_data_structures(int N, int E, Edge* edges, float* k, int* nodes_comm, int* new_nodes_comm, int* c, float* ac)
{
    CUDA_CHECK(hipMemset(k, '\0', N * sizeof(float)));
    prepare_data_structures_kernel<<<BLOCKS, THREADS_PER_BLOCK>>>(N, E, edges, c, k, nodes_comm, new_nodes_comm);
    CUDA_CHECK(hipMemcpy(ac, k, N * sizeof(float), hipMemcpyDeviceToDevice));
}

__device__ uint32_t arr_hash(uint64_t key, int seed, uint64_t N, uint32_t SIZE)
{
    uint64_t l = N * N * seed + key - 1;
    l = (l >> 32) * 1605375019ULL + (l & 0xffffffffULL) * 553437317ULL + 3471094223ULL;
    l = (l >> 32) * 2769702083ULL + (l & 0xffffffffULL) * 3924398899ULL + 2998053229ULL;
    return l & (SIZE - 1);
}

__device__ uint32_t getpos(uint64_t* owner, uint64_t key, int N, uint32_t SIZE)
{
    for (int it = 0; ; ++it)
    {
        uint32_t pos = arr_hash(key, it, N, SIZE);
        if (owner[pos] == key)
        {
            return pos;
        }
        else if (owner[pos] == 0)
        {
            if (atomicCAS((unsigned long long*)&owner[pos], (unsigned long long)(0), (unsigned long long)key) == 0)
            {
                return pos;
            }
            else if (owner[pos] == key)
            {
                return pos;
            }
        }
    }
}

__global__  void compute_changes_kernel(int N, int E, float* changes, uint64_t* owner, Edge* edges, int* c, uint32_t SIZE)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int num_threads = blockDim.x * gridDim.x;
    for (int e = tid; e < E; e += num_threads)
    {
        int vertex = edges[e].src;
        if (edges[e].dst != vertex)
        {
            uint64_t key = (uint64_t)N * vertex + c[edges[e].dst] + 1;
            uint32_t pos = getpos(owner, key, N, SIZE);
            atomicAdd(&changes[pos], edges[e].weight);
        }
    }
}

union Magic
{
    unsigned long long encoded;
    struct {
        int comm;
        float change;
    } decoded;
};

static_assert(sizeof(Magic) == 8, "too much magic");

__global__ void prepare_magic_kernel(int N, Magic* magic, int* c)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int num_threads = blockDim.x * gridDim.x;
    for (int v = tid; v < N; v += num_threads)
    {
        magic[v].decoded.comm = c[v];
        magic[v].decoded.change = 0;
    }
}

__global__ void modularity_optimisation_kernel(int N, int E, Edge* edges, int* c, float* k, int* nodes_comm, int* new_nodes_comm, float* ac, float m, float* changes, uint64_t* owner, Magic* magic, uint32_t SIZE)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int num_threads = blockDim.x * gridDim.x;
    for (int e = tid; e < E; e += num_threads)
    {
        int i = c[edges[e].dst];
        int vertex = edges[e].src;
        if (nodes_comm[c[vertex]] <= 1 && nodes_comm[i] <= 1 && i >= c[vertex])
        {
            continue;
        }
        uint32_t pos1 = getpos(owner, (uint64_t)N * vertex + i + 1, N, SIZE);
        uint32_t pos2 = getpos(owner, (uint64_t)N * vertex + c[vertex] + 1, N, SIZE);
        float change = (changes[pos1] - changes[pos2]) / m + k[vertex] * ((ac[c[vertex]] - k[vertex]) - ac[i]) / (2 * m * m);
        if (change < EPS)
        {
            continue;
        }
        Magic new_magic;
        new_magic.decoded.comm = i;
        new_magic.decoded.change = change;
        while (true)
        {
            Magic local_magic = magic[vertex];
            if ((change > local_magic.decoded.change ||
                    (fabs(change - local_magic.decoded.change) < EPS && i < local_magic.decoded.comm)))
            {

                if (atomicCAS((unsigned long long*)(magic + vertex),
                            local_magic.encoded, new_magic.encoded)
                        == local_magic.encoded)
                {
                    atomicAdd(new_nodes_comm + i, 1);
                    atomicAdd(new_nodes_comm + local_magic.decoded.comm, -1);
                    break;
                }
            }
            else break;
        }
    }
}

__global__ void update_ac_kernel(int N, float* ac, int* c, float* k)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int num_threads = blockDim.x * gridDim.x;
    for (int v = tid; v < N; v += num_threads)
    {
        atomicAdd(&ac[c[v]], k[v]);
    }
}

__global__ void compute_new_c_kernel(int N, Magic* magic, int* new_c)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int num_threads = blockDim.x * gridDim.x;
    for (int v = tid; v < N; v += num_threads)
    {
        new_c[v] = magic[v].decoded.comm;
    }
}

__global__ void debug_kernel_c(int N, int* c)
{
    printf("c in modularity_optimisation: ");
    for (int i = 0; i < N; ++i)
    {
        printf("(%d %d) ", i, c[i]);
    }
    printf("\n");
}

__global__ void debug_kernel_ac(int N, float* ac)
{
    printf("ac in modularity_optimisation: ");
    for (int i = 0; i < N; ++i)
    {
        printf("(%d %f) ", i, ac[i]);
    }
    printf("\n");
}

__global__ void debug_kernel_new_c(int N, int* c)
{
    printf("new_c after modularity_optimisation: ");
    for (int i = 0; i < N; ++i)
    {
        printf("(%d %d) ", i, c[i]);
    }
    printf("\n");
}

__global__ void debug_kernel_nodes_comm(int N, int* c)
{
    printf("(new) nodes_comm: ");
    for (int i = 0; i < N; ++i)
    {
        printf("(%d %d) ", i, c[i]);
    }
    printf("\n");
}

__host__ void modularity_optimisation(int N, int E, Edge* edges, int* c, float* k, int* new_c, int* nodes_comm, int* new_nodes_comm, float* ac, float m, float* changes, uint64_t* owner, Magic* magic)
{
//    printf("N = %d E = %d m = %f\n", N, E, m);
//    debug_kernel_c<<<1, 1>>>(N, c);
//    debug_kernel_ac<<<1, 1>>>(N, ac);
//    debug_kernel_nodes_comm<<<1, 1>>>(N, nodes_comm);
//    debug_kernel_nodes_comm<<<1, 1>>>(N, new_nodes_comm);
    CUDA_CHECK(hipMemset(owner, '\0', sizeof(int) * ARRAY_SIZE));
    CUDA_CHECK(hipMemset(changes, '\0', sizeof(float) * ARRAY_SIZE));
    compute_changes_kernel<<<BLOCKS, THREADS_PER_BLOCK>>>(N, E, changes, owner, edges, c, ARRAY_SIZE);

    prepare_magic_kernel<<<BLOCKS, THREADS_PER_BLOCK>>>(N, magic, c);
    modularity_optimisation_kernel<<<BLOCKS, THREADS_PER_BLOCK>>>(N, E, edges, c, k, nodes_comm, new_nodes_comm, ac, m, changes, owner, magic, ARRAY_SIZE);
    compute_new_c_kernel<<<BLOCKS, THREADS_PER_BLOCK>>>(N, magic, new_c);
    CUDA_CHECK(hipMemset(ac, '\0', N * sizeof(float)));
    update_ac_kernel<<<BLOCKS, THREADS_PER_BLOCK>>>(N, ac, new_c, k);
//    debug_kernel_new_c<<<1, 1>>>(N, new_c);
}

__global__ void compute_modularity_kernel(int N, int E, Edge* edges, int* c, float* changes, float* ac, float* ac_helper)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int num_threads = blockDim.x * gridDim.x;
    for (int i = tid; i < E; i+= num_threads)
    {
        if (c[edges[i].src] == c[edges[i].dst])
        {
            atomicAdd(changes + edges[i].src, edges[i].weight);
        }
    }
    for (int i = tid; i < N; i += num_threads)
    {
        ac_helper[i] = ac[i] * ac[i];
    }
}

__global__ void debug_kernel(int N, int* new_c)
{
    printf("new c: ");
    for (int i = 0; i < N; ++i)
    {
        printf("(%d %d) ", i, new_c[i]);
    }
    printf("\n");
}

__global__ void debug_kernel_2(int N, float* ac)
{
    for (int i = 0; i < N; ++i)
        printf("ac[%d] = %f ", i, ac[i]);
    printf("\n");
}

__host__ float compute_modularity(int N, int E, Edge* edges, int* c, float* changes, float* ac, float* ac_helper, float m)
{
//    debug_kernel<<<1, 1>>>(N, c);
//    debug_kernel_2<<<1, 1>>>(N, ac);
    CUDA_CHECK(hipMemset(changes, 0, N * sizeof(float)));

    compute_modularity_kernel<<<BLOCKS, THREADS_PER_BLOCK>>>(N, E, edges, c, changes, ac, ac_helper);

    float q1 = thrust::reduce(thrust::device, changes, changes + N);
    float q2 = thrust::reduce(thrust::device, ac_helper, ac_helper + N);

    return q1 / (2 * m) - q2 / (4 * m * m);
}

__global__ void prepare_reorder_kernel(int N, int* reorder, int* c)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int num_threads = blockDim.x * gridDim.x;
    for (int i = tid; i < N; i += num_threads)
        reorder[c[i]] = 1;
}

__global__ void aggregate_kernel(int E, int orig_N, Edge* edges, int* reorder, int* c, int* final_communities)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int num_threads = blockDim.x * gridDim.x;
    for (int i = tid; i < E; i += num_threads)
    {
        edges[i].src = reorder[c[edges[i].src]];
        edges[i].dst = reorder[c[edges[i].dst]];
    }
    for (int i = tid; i < orig_N; i += num_threads)
    {
        final_communities[i] = reorder[c[final_communities[i]]];
    }
}

__host__ void aggregate(int& N, int E, int orig_N, Edge* edges, int* c, int* final_communities, float* k, int* nodes_comm, int* new_nodes_comm, float* ac)
{
    int* reorder;
    CUDA_CHECK(hipMalloc((void**)&reorder, (N + 1) * sizeof(int)));
    CUDA_CHECK(hipMemset(reorder, 0, (N + 1) * sizeof(int)));
    prepare_reorder_kernel<<<BLOCKS, THREADS_PER_BLOCK>>>(N, reorder, c);
    thrust::exclusive_scan(thrust::device, reorder, reorder + N + 1, reorder);
    aggregate_kernel<<<BLOCKS, THREADS_PER_BLOCK>>>(E, orig_N, edges, reorder, c, final_communities);
    N = device_fetch_var(reorder + N);
    CUDA_CHECK(hipFree(reorder));
    prepare_data_structures(N, E, edges, k, nodes_comm, new_nodes_comm, c, ac);
}

__global__ void prepare_final_communities(int* fc, int N)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int num_threads = blockDim.x * gridDim.x;
    for (int i = tid; i < N; i += num_threads)
    {
        fc[i] = i;
    }
}

void gpu_louvain(int N_, Edge* edges_, int E_, float min_gain, bool verbose, std::map<int, int>& reorder)
{
    int N;
    int orig_N;
    int E;
    Edge* edges;
    Edge* orig_edges;
    float m = 0;
    int* final_communities;
    int* c;
    int* new_c;
    float* k;
    float* ac;
    float* ac_helper;
    float* changes;
    uint64_t* owner;
    int* nodes_comm;
    int* new_nodes_comm;
    Magic* magic;

    N = N_;
    E = E_;
    orig_N = N_;
    orig_edges = edges_;

    while (E * 10 < ARRAY_SIZE)
    {
        ARRAY_SIZE >>= 1;
    }

    CUDA_CHECK(hipMalloc((void**)&final_communities, N * sizeof(int)));
    prepare_final_communities<<<BLOCKS, THREADS_PER_BLOCK>>>(final_communities, N);
    CUDA_CHECK(hipMalloc((void**)&c, N * sizeof(int)));
    CUDA_CHECK(hipMalloc((void**)&new_c, N * sizeof(int)));
    CUDA_CHECK(hipMalloc((void**)&nodes_comm, N * sizeof(int)));
    CUDA_CHECK(hipMalloc((void**)&new_nodes_comm, N * sizeof(int)));
    CUDA_CHECK(hipMalloc((void**)&k, N * sizeof(float)));
    CUDA_CHECK(hipMalloc((void**)&ac, N * sizeof(float)));
    CUDA_CHECK(hipMalloc((void**)&ac_helper, N * sizeof(float)));
    CUDA_CHECK(hipMalloc((void**)&edges, sizeof(Edge) * E));
    CUDA_CHECK(hipMalloc((void**)&changes, ARRAY_SIZE * sizeof(float)));
    CUDA_CHECK(hipMalloc((void**)&owner, ARRAY_SIZE * sizeof(uint64_t)));
    CUDA_CHECK(hipMalloc((void**)&magic, N * sizeof(Magic)));
    CUDA_CHECK(hipMemcpy(edges, orig_edges, sizeof(Edge) * E, hipMemcpyHostToDevice));

    prepare_data_structures(N, E, edges, k, nodes_comm, new_nodes_comm, c, ac);
    // we can compute it on cpu because it's done only once
    for (int i = 0; i < E; ++i)
    {
        m += orig_edges[i].weight;
    }
    m /= 2;

    float old_modularity = compute_modularity(N, E, edges, c, changes, ac, ac_helper, m), new_modularity = 0, sum = 0;
    do
    {
        sum = 0;
        do
        {
            modularity_optimisation(N, E, edges, c, k, new_c, nodes_comm, new_nodes_comm, ac, m, changes, owner, magic);
            new_modularity = compute_modularity(N, E, edges, new_c, changes, ac, ac_helper, m);
//            printf("modularity_change = %.9f\n", new_modularity - old_modularity);
            if (new_modularity - old_modularity > EPS)
            {
                std::swap(c, new_c);
                sum += new_modularity - old_modularity;
                std::swap(new_modularity, old_modularity);
                CUDA_CHECK(hipMemcpy(nodes_comm, new_nodes_comm, N * sizeof(int), hipMemcpyDeviceToDevice));
            }
            else break;
        } while (true);
//        printf("aggregating\n");
        aggregate(N, E, orig_N, edges, c, final_communities, k, nodes_comm, new_nodes_comm, ac);
    } while (sum > min_gain);

    int* final_communities_host = (int*)malloc(orig_N * sizeof(int));
    CUDA_CHECK(hipMemcpy(final_communities_host, final_communities, orig_N * sizeof(int), hipMemcpyDeviceToHost));

    float* k_host = (float*)malloc(orig_N * sizeof(float));
    memset(k_host, '\0', orig_N * sizeof(float));
    for (int i = 0; i < E; ++i)
    {
        k_host[orig_edges[i].dst] += orig_edges[i].weight;
    }
    float* ac_host = (float*)malloc(orig_N * sizeof(float));
    memset(ac_host, '\0', orig_N * sizeof(float));
    CUDA_CHECK(hipDeviceSynchronize());
    for (int i = 0; i < orig_N; ++i)
    {
        ac_host[final_communities_host[i]] += k_host[i];
    }
    float* e_host = (float*)malloc(orig_N * sizeof(float));
    memset(e_host, '\0', orig_N * sizeof(float));
    for (int i = 0; i < E; ++i)
    {
        if (final_communities_host[orig_edges[i].src] == final_communities_host[orig_edges[i].dst])
        {
            e_host[orig_edges[i].src] += orig_edges[i].weight;
        }
    }
    float q = 0;
    for (int i = 0; i < orig_N; ++i)
    {
        q += e_host[i] / (2 * m);
    }
    for (int i = 0; i < orig_N; ++i)
    {
        q -= ac_host[i] * ac_host[i] / (4 * m * m);
    }

    printf("%f\n", q);
    printf("0 0\n"); // TODO measure times
    if (verbose)
    {
        printf("%d\n", N);
        for (int i = 0; i < N; ++i)
        {
            printf("%d ", i + 1);
            for (int v = 0; v < orig_N; ++v)
            {
                if (final_communities_host[v] == i)
                    printf("%d ", reorder[v]);
            }
            printf("\n");
        }
    }

    free(final_communities_host);
    free(k_host);
    free(ac_host);
    free(e_host);
    CUDA_CHECK(hipFree(final_communities));
    CUDA_CHECK(hipFree(c));
    CUDA_CHECK(hipFree(new_c));
    CUDA_CHECK(hipFree(k));
    CUDA_CHECK(hipFree(ac));
    CUDA_CHECK(hipFree(ac_helper));
    CUDA_CHECK(hipFree(changes));
    CUDA_CHECK(hipFree(owner));
    CUDA_CHECK(hipFree(nodes_comm));
    CUDA_CHECK(hipFree(new_nodes_comm));
    CUDA_CHECK(hipFree(edges));
    CUDA_CHECK(hipFree(magic));
}


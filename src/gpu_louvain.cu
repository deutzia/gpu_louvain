#include "hip/hip_runtime.h"
#include "cpu_louvain.h"

#include <algorithm>
#include <stdint.h>
#include <string.h>
#include <thrust/sort.h>
#include <thrust/scan.h>

#include "utils.h"

#define BLOCKS 80
#define THREADS_PER_BLOCK 128
#define ARRAY_SIZE (1 << 28)

struct edge_cmp
{
    __device__ bool operator()(const Edge& a, const Edge& b)
    {
        return a.src < b.src;
    }
};

__global__ void
prepare_data_structures_kernel1(int N, int E, int* degrees, Edge* edges, int* c, float* k, int* order, int* nodes_comm)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int num_threads = blockDim.x * gridDim.x;
    for (int i = tid; i < E; i += num_threads)
    {
        atomicAdd(&degrees[edges[i].src], 1);
    }
    for (int i = tid; i < N; i += num_threads)
    {
        c[i] = i;
    }
    for (int i = tid; i < E; i += num_threads)
    {
        atomicAdd(&k[edges[i].dst], edges[i].weight);
    }
    for (int i = tid; i < N; i += num_threads)
    {
        order[i] = i;
    }

    for (int i = tid; i < N; i += num_threads)
    {
        nodes_comm[i] = 1;
    }
}

__global__ void prepare_data_structures_kernel2(int N, int E, Edge* edges, int* e_start, int* e_end)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int num_threads = blockDim.x * gridDim.x;
    for (int i = tid; i < E; i += num_threads)
    {
        if (i == 0 || edges[i].src != edges[i-1].src)
        {
            e_start[edges[i].src] = i;
        }
        if (i == E - 1 || edges[i].src != edges[i+1].src)
        {
            e_end[edges[i].src] = i + 1;
        }
    }
}

struct vertex_cmp
{
    int* degrees;
    vertex_cmp(int* d)
    : degrees(d)
    {}
    __device__ bool operator()(int a, int b)
    {
        return degrees[a] < degrees[b];
    }
};

__host__ void prepare_data_structures(int N, int E, Edge* edges, int* degrees, int* e_start, int* e_end, float* k, int* order, int* nodes_comm, int* c, float* ac)
{
    thrust::sort(thrust::device, edges, edges + E, edge_cmp());
    CUDA_CHECK(hipMemset(degrees, 0, N * sizeof(int)));
    CUDA_CHECK(hipMemset(e_start, '\0', N * sizeof(int)));
    CUDA_CHECK(hipMemset(e_end, '\0', N * sizeof(int)));
    CUDA_CHECK(hipMemset(k, '\0', N * sizeof(float)));
    prepare_data_structures_kernel1<<<BLOCKS, THREADS_PER_BLOCK>>>(N, E, degrees, edges, c, k, order, nodes_comm);
    prepare_data_structures_kernel2<<<BLOCKS, THREADS_PER_BLOCK>>>(N, E, edges, e_start, e_end);
    CUDA_CHECK(hipMemcpy(ac, k, N * sizeof(float), hipMemcpyDeviceToDevice));
    thrust::sort(thrust::device, order, order + N, vertex_cmp(degrees));
}

__device__ uint32_t arr_hash(uint64_t key, int seed, uint64_t N)
{
    uint64_t l = N * N * seed + key - 1;
    l = (l << 32) * 1605375019ULL + (l & 0xffffffffULL) * 553437317ULL + 3471094223ULL;
    l = (l << 32) * 2769702083ULL + (l & 0xffffffffULL) * 3924398899ULL + 2998053229ULL;
    return l % ARRAY_SIZE;
}

__device__ uint32_t getpos(uint64_t* owner, uint64_t key, int N)
{
    for (int it = 0; ; ++it)
    {
        uint32_t pos = arr_hash(key, it, N);
        if (owner[pos] == key)
        {
            return pos;
        }
        else if (owner[pos] == 0)
        {
            if (atomicCAS((unsigned long long*)&owner[pos], (unsigned long long)(0), (unsigned long long)key) == 0)
            {
                return pos;
            }
            // TODO if there's more than one thread for each vertex this breaks
        }
    }
}

__device__ float compute_move(int vertex, int N, float* changes, uint64_t* owner, int* e_start, int* e_end, Edge* edges, int* c, float* k, int* new_c, int* nodes_comm, int* new_nodes_comm, float* ac, float m)
{
    for (int j = e_start[vertex]; j < e_end[vertex]; ++j)
    {
        if (edges[j].dst != vertex)
        {
            uint64_t key = (uint64_t)N * vertex + c[edges[j].dst] + 1;
            uint32_t pos = getpos(owner, key, N);
            atomicAdd(&changes[pos], edges[j].weight);
        }
    }
    int resultComm = c[vertex];
    float resultChange = 0;
    for (int e = e_start[vertex]; e < e_end[vertex]; ++e)
    {
        int i = c[edges[e].dst];
        uint32_t pos1 = getpos(owner, (uint64_t)N * vertex + i + 1, N);
        uint32_t pos2 = getpos(owner, (uint32_t)N * vertex + c[vertex] + 1, N);
        float change = 1 / m * (changes[pos1] - changes[pos2]) + k[vertex] * ((ac[c[vertex]] - k[vertex]) - ac[i]) / (2 * m * m);
        if ((change > resultChange || (change == resultChange && i < resultComm)) &&
                (nodes_comm[c[vertex]] > 1 ||
                 nodes_comm[i] > 1 ||
                 i < c[vertex]))
        {
            new_nodes_comm[i]++;
            new_nodes_comm[resultComm]--;
            resultChange = change;
            resultComm = i;
        }
    }
    new_c[vertex] = resultComm;
    return resultChange;
}

__global__ void modularity_optimisation_kernel(int N, int* e_start, int* e_end, Edge* edges, int* c, float* k, int* new_c, int* nodes_comm, int* new_nodes_comm, float* ac, float m, float* gain, float* changes, uint64_t* owner, int* order)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int num_threads = blockDim.x * gridDim.x;
    for (int v = tid; v < N; v += num_threads)
    {
        atomicAdd(gain, compute_move(order[v], N, changes, owner, e_start, e_end, edges, c, k, new_c, nodes_comm, new_nodes_comm, ac, m));
    }
}

__global__ void update_ac_kernel(int N, float* ac, int* c, float* k)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int num_threads = blockDim.x * gridDim.x;
    for (int v = tid; v < N; v += num_threads)
    {
        atomicAdd(&ac[c[v]], k[v]);
    }
}

// return modularity gain
__host__ float modularity_optimisation(int N, int* e_start, int* e_end, Edge* edges, int* c, float* k, int* new_c, int* nodes_comm, int* new_nodes_comm, float* ac, float m, float* changes, uint64_t* owner, int* order)
{
    float* gain;
    CUDA_CHECK(hipMalloc((void**)&gain, sizeof(float)));
    CUDA_CHECK(hipMemset(gain, '\0', sizeof(float)));
    CUDA_CHECK(hipMemset(owner, '\0', sizeof(int) * ARRAY_SIZE));
    CUDA_CHECK(hipMemset(changes, '\0', sizeof(float) * ARRAY_SIZE));
    modularity_optimisation_kernel<<<BLOCKS, THREADS_PER_BLOCK>>>(N, e_start, e_end, edges, c, k, new_c, nodes_comm, new_nodes_comm, ac, m, gain, changes, owner, order);
    std::swap(c, new_c);
    CUDA_CHECK(hipMemcpy(nodes_comm, new_nodes_comm, N * sizeof(int), hipMemcpyDeviceToDevice));
    CUDA_CHECK(hipMemset(ac, '\0', N * sizeof(float)));
    update_ac_kernel<<<BLOCKS, THREADS_PER_BLOCK>>>(N, ac, c, k);
    float result = device_fetch_var(gain);
    CUDA_CHECK(hipFree(gain));
    return result;
}

__global__ void prepare_reorder_kernel(int N, int* reorder, int* c)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int num_threads = blockDim.x * gridDim.x;
    for (int i = tid; i < N; i += num_threads)
        reorder[c[i]] = 1;
}

__global__ void aggregate_kernel(int E, int orig_N, Edge* edges, int* reorder, int* c, int* final_communities)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int num_threads = blockDim.x * gridDim.x;
    for (int i = tid; i < E; i += num_threads)
    {
        edges[i].src = reorder[c[edges[i].src]];
        edges[i].dst = reorder[c[edges[i].dst]];
    }
    for (int i = tid; i < orig_N; i += num_threads)
    {
        final_communities[i] = reorder[c[final_communities[i]]];
    }
}

__host__ void aggregate(int& N, int E, int orig_N, Edge* edges, int* c, int* final_communities, int* degrees, int* e_start, int* e_end, float* k, int* order, int* nodes_comm, float* ac)
{
    int* reorder;
    CUDA_CHECK(hipMalloc((void**)&reorder, (N + 1) * sizeof(int)));
    CUDA_CHECK(hipMemset(reorder, 0, (N + 1) * sizeof(int)));
    prepare_reorder_kernel<<<BLOCKS, THREADS_PER_BLOCK>>>(N, reorder, c);
    thrust::exclusive_scan(thrust::device, reorder, reorder + N + 1, reorder);
    aggregate_kernel<<<BLOCKS, THREADS_PER_BLOCK>>>(E, orig_N, edges, reorder, c, final_communities);
    N = device_fetch_var(reorder + N);
    CUDA_CHECK(hipFree(reorder));
    prepare_data_structures(N, E, edges, degrees, e_start, e_end, k, order, nodes_comm, c, ac);
}

__global__ void prepare_final_communities(int* fc, int N)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int num_threads = blockDim.x * gridDim.x;
    for (int i = tid; i < N; i += num_threads)
    {
        fc[i] = i;
    }
}

void gpu_louvain(int N_, Edge* edges_, int E_, float min_gain, bool verbose)
{
    int N;
    int orig_N;
    int E;
    Edge* edges;
    Edge* orig_edges;
    float m;
    int* e_start;
    int* e_end;
    int* degrees;
    int* final_communities;
    int* c;
    int* new_c;
    float* k;
    float* ac;
    float* changes;
    uint64_t* owner;
    int* order;
    int* nodes_comm;
    int* new_nodes_comm;

    N = N_;
    E = E_;
    orig_N = N_;
    orig_edges = edges_;

    CUDA_CHECK(hipMalloc((void**)&final_communities, N * sizeof(int)));
    prepare_final_communities<<<BLOCKS, THREADS_PER_BLOCK>>>(final_communities, N);
    CUDA_CHECK(hipMalloc((void**)&degrees, N * sizeof(int)));
    CUDA_CHECK(hipMalloc((void**)&e_start, N * sizeof(int)));
    CUDA_CHECK(hipMalloc((void**)&e_end, N * sizeof(int)));
    CUDA_CHECK(hipMalloc((void**)&c, N * sizeof(int)));
    CUDA_CHECK(hipMalloc((void**)&new_c, N * sizeof(int)));
    CUDA_CHECK(hipMalloc((void**)&order, N * sizeof(int)));
    CUDA_CHECK(hipMalloc((void**)&nodes_comm, N * sizeof(int)));
    CUDA_CHECK(hipMalloc((void**)&new_nodes_comm, N * sizeof(int)));
    CUDA_CHECK(hipMalloc((void**)&k, N * sizeof(float)));
    CUDA_CHECK(hipMalloc((void**)&ac, N * sizeof(float)));
    CUDA_CHECK(hipMalloc((void**)&edges, sizeof(Edge) * E));
    CUDA_CHECK(hipMalloc((void**)&changes, ARRAY_SIZE * sizeof(float)));
    CUDA_CHECK(hipMalloc((void**)&owner, ARRAY_SIZE * sizeof(uint64_t)));
    CUDA_CHECK(hipMemcpy(edges, orig_edges, sizeof(Edge) * E, hipMemcpyHostToDevice));

    prepare_data_structures(N, E, edges, degrees, e_start, e_end, k, order, nodes_comm, c, ac);
    // we can compute it on cpu because it's done only once
    for (int i = 0; i < E; ++i)
    {
        m += orig_edges[i].weight;
    }
    m /= 2;
    float modularity_change = 0;
    do
    {
        modularity_change = modularity_optimisation(N, e_start, e_end, edges, c, k, new_c, nodes_comm, new_nodes_comm, ac, m, changes, owner, order);
        std::swap(c, new_c);
        aggregate(N, E, orig_N, edges, c, final_communities, degrees, e_start, e_end, k, order, nodes_comm, ac);
    } while (modularity_change > min_gain);

    int* final_communities_host = (int*)malloc(orig_N * sizeof(int));
    CUDA_CHECK(hipMemcpy(final_communities_host, final_communities, orig_N * sizeof(int), hipMemcpyDeviceToHost));

    float* k_host = (float*)malloc(orig_N * sizeof(float));
    memset(k_host, '\0', orig_N * sizeof(float));
    for (int i = 0; i < E; ++i)
    {
        k_host[orig_edges[i].dst] += orig_edges[i].weight;
    }
    float* ac_host = (float*)malloc(orig_N * sizeof(float));
    memset(ac_host, '\0', orig_N * sizeof(float));
    hipDeviceSynchronize();
    for (int i = 0; i < orig_N; ++i)
    {
        ac_host[final_communities_host[i]] += k_host[i];
    }
    float* e_host = (float*)malloc(orig_N * sizeof(float));
    memset(e_host, '\0', orig_N * sizeof(float));
    for (int i = 0; i < E; ++i)
    {
        if (final_communities_host[orig_edges[i].src] == final_communities_host[orig_edges[i].dst])
        {
            e_host[orig_edges[i].src] += orig_edges[i].weight;
        }
    }
    float q = 0;
    for (int i = 0; i < orig_N; ++i)
    {
        q += e_host[i] / (2 * m);
    }
    for (int i = 0; i < orig_N; ++i)
    {
        q -= ac_host[i] * ac_host[i] / (4 * m * m);
    }

    printf("%f\n", q);
    printf("0 0\n"); // TODO measure times
    printf("%d\n", N);
    if (verbose)
    {
        for (int i = 0; i < N; ++i)
        {
            printf("%d ", i + 1);
            for (int v = 0; v < orig_N; ++v)
            {
                if (final_communities_host[v] == i)
                    printf("%d ", v + 1);
            }
            printf("\n");
        }
    }

    free(final_communities_host);
    free(k_host);
    free(ac_host);
    free(e_host);
    CUDA_CHECK(hipFree(final_communities));
    CUDA_CHECK(hipFree(c));
    CUDA_CHECK(hipFree(new_c));
    CUDA_CHECK(hipFree(k));
    CUDA_CHECK(hipFree(ac));
    CUDA_CHECK(hipFree(order));
    CUDA_CHECK(hipFree(degrees));
    CUDA_CHECK(hipFree(e_start));
    CUDA_CHECK(hipFree(e_end));
    CUDA_CHECK(hipFree(changes));
    CUDA_CHECK(hipFree(owner));
    CUDA_CHECK(hipFree(nodes_comm));
    CUDA_CHECK(hipFree(new_nodes_comm));
    CUDA_CHECK(hipFree(edges));
}

